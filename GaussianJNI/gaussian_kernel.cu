
#include <hip/hip_runtime.h>
//include "gaussian.h"
#define BLOCK_SIZE 16

__global__ void
Gaussian_CUDA(int* A, int wA, int row, int* temp)
{
	int bx, tx, by, ty;

	// Block index
	//MODIFY HERE to get your block indexes
	bx=blockIdx.x;
	by=blockIdx.y;
	

	// Thread index
	//MODIFY HERE to get your thread indexes
	tx=threadIdx.x;
	ty=threadIdx.y;

	int index = wA * BLOCK_SIZE * by + BLOCK_SIZE * bx + wA * ty + tx;
    
	int index_col = BLOCK_SIZE * bx + tx;
	int index_row = BLOCK_SIZE * by + ty;

//	int i;
	__shared__ int s;

/*	
	if(index_row==row&&index_col==row)
	{
		for(i=row+1;i<wA;i++)
			temp[i]=A[(i*wA)+row]/A[(row*wA)+row];
	}
	__syncthreads();


	if(index_row>row)
		A[(index_row*wA)+index_col]-=(temp[(index_row)]*A[(row*wA)+index_col]);
	__syncthreads();	
	
*/
	if(index_row==row)
		temp[index_col]=A[(index_row*wA)+index_col];
	__syncthreads();

	if(index_row>row)
	{
		s=A[(index_row*wA)+row]/temp[row];
		__syncthreads();
		A[index]-=A[(row*wA)+index_col]*s;
	}
	__syncthreads();
	
}

