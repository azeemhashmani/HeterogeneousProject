// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "gaussian_kernel.cu"
#include <jni.h>
#include "Gaussian.h"
#define OUTPUT

void runTest(int argc, char** argv);

double gettime() {
	struct timeval t;
	gettimeofday(&t,NULL);
	return t.tv_sec+t.tv_usec*1e-6;
}

int
main(int argc, char** argv)
{
	//runTest(argc, argv);
}

//void runTest(int h_A, char** argv)
JNIEXPORT jint JNICALL Java_Gaussian_runTest
  (JNIEnv *env, jobject j_obj, jintArray j_A, jint dim)
{
	hipError_t err;
    
    //display the test case
	/*
	for ( int m = 0 ; m < dim; m++){
		for ( int n = 0 ; n < dim; n++){
			printf("%d ", h_A[m * dim + n]);
		}
		printf("\n");
	}
	*/
	
	unsigned int size_A = dim * dim;
	unsigned int mem_size_A = sizeof(int) * size_A;
	printf("Inside CUDA code\n");
	jint *h_A = env->GetIntArrayElements(j_A, 0);

    // allocate device memory for the matrix A
	int* d_A;
	hipMalloc((void**)&d_A,mem_size_A);
	
	//MODIFY HERE 
    
	int* temp; //temporary array to store dim number of integer elements
	//MODIFY HERE to allocate memory for temp array 
	//temp=(int*)malloc(dim*sizeof(int));
	hipMalloc((void**)&temp,dim*sizeof(int));
	
	// copy host memory to device
	double timer1 = gettime();
	//MODIFY HERE Copy the Matrix A to GPU memory
	if((err=hipMemcpy((void*)d_A,(void*)h_A,mem_size_A,hipMemcpyHostToDevice))!=hipSuccess)
		printf("Error: Host to Device copy%d\n",err);

	// setup execution parameters
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(dim / threads.x, dim / threads.y);

	// execute the kernel
	for ( int i = 0 ; i < dim ; i++){
		Gaussian_CUDA<<< grid, threads >>>(d_A, dim, i, temp);
	}

	// copy result from device to host
	//MODIFY HERE
	if((err=hipMemcpy((void*)h_A,(void*)d_A,mem_size_A, hipMemcpyDeviceToHost))!=hipSuccess)
		printf("Error:Device to Device copy%d\n",err);

	

	double timer2 = gettime();
	printf("GPU time = %lf\n",(timer2-timer1)*1000);

#ifdef OUTPUT

	//the result should be I(dim*dim)
	for ( int m = 0 ; m < dim; m++){
		for ( int n = 0 ; n < dim; n++){
			printf("%d ", h_A[m * dim + n]);
		}
		printf("\n");
	}
#endif

	env->ReleaseIntArrayElements(j_A, h_A, 0);
	//free(h_A);
	hipFree(d_A);
	hipFree(temp);
	return 0;
}
