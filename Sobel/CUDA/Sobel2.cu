#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>


#define SWAP(a,b) tempr=(a);(a)=(b);(b)=tempr

#define DEF_IN "mri" //name of default input image file
#define DEF_OUT "part1_edge" //name of default filtered output image

C
C
#define DEF_BINARY "part1_binary_edge"
#define def_xsize 256    //default X axis size
#define def_ysize 256    //default Y axis size
#define BLOCK_SIZE 16	//block size for parallel gpu


double gettime() {
        struct timeval t;
        gettimeofday(&t,NULL);
        return t.tv_sec+t.tv_usec*1e-6;
}


__global__ void mykernel(float *temp2,float *out3,int xsize,int ysize,float *outH, float *outV)
{
	int bx, tx, by, ty;

        // Block index
        bx=blockIdx.x;
        by=blockIdx.y;


        // Thread index
        tx=threadIdx.x;
        ty=threadIdx.y;

        //int index = wA * BLOCK_SIZE * by + BLOCK_SIZE * bx + wA * ty + tx;

        int index_col = BLOCK_SIZE * bx + tx;
        int index_row = BLOCK_SIZE * by + ty;
	
	int k,l;
	float sobelH[9]={-1,-2,-1,0,0,0,1,2,1};
        float sobelV[9]={-1,0,1,-2,0,2,-1,0,1};


	for(k=-1;k<=1;k++)
	for(l=-1;l<=1;l++)
	{	
		outH[(index_row*xsize)+index_col]+=(temp2[((index_row+1+k)*(xsize+2))+(index_col+1+l)]*sobelH[(k+1)*3+(l+1)]);
		outV[(index_row*xsize)+index_col]+=(temp2[((index_row+1+k)*(xsize+2))+(index_col+1+l)]*sobelV[(k+1)*3+(l+1)]);
	}

	out3[(index_row*xsize)+index_col]=abs(outH[(index_row*xsize)+index_col])+abs(outV[(index_row*xsize)+index_col]);
	
	//__syncthreads();
}


void norm_output(float *output[],int xsize,int ysize, FILE *fp_out)
{
    int i,j;
    float max,min;
    unsigned char pixel;
   
    min=output[0][0];
    max=output[0][0];

    for(i=0;i<ysize;i++)
    for(j=0;j<xsize;j++)
    {
        if(output[i][j]<min)
            min=output[i][j];
        if(output[i][j]>max)
            max=output[i][j];
    }
   
    //printf("min is %f and max is %f\n",min,max);
    for(i=0;i<ysize;i++)
    for(j=0;j<xsize;j++)
    {
        output[i][j]=((output[i][j]-min)/(max-min))*255;
        pixel=(unsigned char)output[i][j];
	fwrite(&pixel,sizeof(char),1,fp_out);
    }
}
 
void sobel_Filter(float *sobel[], int xsize, int ysize)
{
	int i,j;
	
	float *temp;
	float *out1;

	float *d_temp1;
	float *d_out1;

	
	float *outH,*outV;
	hipError_t err;	
	
	if((err=hipMalloc((void**)&outH,(ysize*xsize)*sizeof(float)))!=hipSuccess)
		printf("Malloc outH %d\n",err);
	hipMemset(outH,0,(ysize*xsize)*sizeof(float));
	
	if((err=hipMalloc((void**)&outV,(ysize*xsize)*sizeof(float)))!=hipSuccess)
		printf("Malloc outV %d\n",err);
	hipMemset(outV,0,(ysize*xsize)*sizeof(float));
	

    	temp=(float *)calloc((ysize+2)*(xsize+2),sizeof(float));


	out1=(float *)calloc((ysize*xsize),sizeof(float));
   
	hipMalloc((void**)&d_out1,(xsize)*sizeof(float)*ysize);
	
	if((err=hipMalloc((void**)&d_temp1,(xsize+2)*sizeof(float)*(ysize+2)))!=hipSuccess)
		printf("Malloc out1 %d\n",err);

	for(i=0;i<(ysize);i++)
	for(j=0;j<(xsize);j++)
		temp[((i+1)*(xsize+2))+j+1]=sobel[i][j];
	
	/*	
	for(i=0;i<(ysize);i++)
        {
		for(j=0;j<(xsize);j++)
		{
			printf("%f  ",temp[((i+1)*(xsize+2))+j+1]);
		}
		printf("\n\n");
	}*/

	if((err=hipMemcpy(d_temp1,temp,(xsize+2)*(ysize+2)*sizeof(float),hipMemcpyHostToDevice))!=hipSuccess)
		printf("Copy Host to Device %d\n",err);

	double timer1 = gettime();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
     	dim3 grid((xsize) / threads.x, (xsize) / threads.y);


	mykernel<<<grid,threads>>>(d_temp1,d_out1,xsize,ysize,outH,outV);
	
	if((err=hipMemcpy(out1,d_out1,xsize*sizeof(float)*ysize,hipMemcpyDeviceToHost))!=hipSuccess)
		printf("Copy Device to Host %d\n",err);
       
	double timer2 = gettime();
        printf("\n\nGPU time = %lf\n",(timer2-timer1)*1000);
	
	for(i=0;i<(ysize);i++)
        {
		for(j=0;j<(xsize);j++)
		{
                	sobel[i][j]=out1[(i*xsize)+j];
			//sobel[i][j]=temp[((i+1)*(xsize+2))+j+1];
			//printf("%f   ",sobel[i][j]);
		}
	//	printf("\n");
	}

	free(out1);
	hipFree(d_out1);
	hipFree(outH);
	hipFree(outV);
	hipFree(d_temp1);
}				
			
int main(int argc,char *argv[])
{
    int xsize=def_xsize,ysize=def_ysize,i,j;
    unsigned char pixel;
    unsigned char **data;
    float **sobel;
    char fn_inp[20]=DEF_IN;
    char fn_out[20]=DEF_OUT;
    FILE *fp_inp;
    FILE *fp_out;

    if(argc==4)    //check if arguments are there
    {
        strcpy(fn_inp,argv[1]);    //Take input file name from arg1
        xsize=atoi(argv[2]);    //Take X-Size from arg2
        ysize=atoi(argv[3]);    //Take Y-Size from arg3
    }
    else
    {/*
	printf("\nNo Command line arguments entered.\n");
	printf("Taking Default Parameters as\n");
	printf("Default input image file : 'mri' of size 256*256\n"); */
    }
    //printf("Output file for sobel output of image: %s\n",fn_out);
   
    //Allocate 2d array memory to process image
    data=(unsigned char **)malloc(sizeof(unsigned char*)*ysize);
    for(i=0;i<ysize;i++)
        data[i]=(unsigned char *)malloc(sizeof(unsigned char)*xsize);

    //open input image
    if((fp_inp=fopen(fn_inp,"r"))==NULL) exit(0);
    for(i=0;i<ysize;i++)
    for(j=0;j<xsize;j++)
    {
        //read pixel by pixel and store it in array data
        fread(&pixel,sizeof(char),1,fp_inp);
        data[i][j]=pixel;
    }
    fclose(fp_inp);   
    sobel=(float **)malloc(sizeof(float *)*ysize);
        for(i=0;i<ysize;i++)
                sobel[i]=(float *)malloc(sizeof(float)*xsize);
 

    for(i=0;i<ysize;i++)
    for(j=0;j<xsize;j++)
    {   
        pixel=data[i][j];
        sobel[i][j]=(float)pixel;
    }
    sobel_Filter(sobel,xsize,ysize);
 
    if((fp_out=fopen(fn_out,"w"))==NULL) exit(0);
    norm_output(sobel,xsize,ysize,fp_out);
    fclose(fp_out);

    return(0);    
}

